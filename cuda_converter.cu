#include "hip/hip_runtime.h"
//先测试竖直黑线 水平模板
#include "cuda_converter.h"
#include <stdio.h>
#include <stdlib.h>
#include <hip/hip_runtime.h>
#include ""
#include "hip/device_functions.h"


#define BLOCKSIZE1 64
#define BLOCKSIZE2 48
#define BLOCKSIZE3 16
#define WIDTH      720
#define HEIGHT     576
typedef unsigned char     BYTE;
typedef unsigned int      uint;



cuda_converter::cuda_converter(void)
{
}

cuda_converter::~cuda_converter(void)
{
}
int cuda_converter::InitCUDA()
{
	/************************************************************************/
	/* Init CUDA                                                            */
	/************************************************************************/

	int count = 0;
	int i = 0;

	hipGetDeviceCount(&count);
	if (count == 0) {
		fprintf(stderr, "There is no device.\n");
		return false;
	}

	for (i = 0; i < count; i++) {
		hipDeviceProp_t prop;
		if (hipGetDeviceProperties(&prop, i) == hipSuccess) {
			if (prop.major >= 1) {
				break;
			}
		}
	}
	if (i == count) {
		fprintf(stderr, "There is no device supporting CUDA.\n");
		return false;
	}
	hipSetDevice(i);

	printf("CUDA initialized.\n");
	return true;

}


int cuda_converter::TranslateOutput(int width, int height, unsigned char *out_data, unsigned char *ou_data)  //out_data:d_result   直接输出四个独立的二值化标记图
{
	int size = width * height;
	hipMemcpy(out_data, d_blackver, sizeof(unsigned char)*size, hipMemcpyDeviceToHost);
	hipMemcpy(ou_data, d_blackhor, sizeof(unsigned char)*size, hipMemcpyDeviceToHost);
	return 1;
}

int cuda_converter::TranslateOutputB(int width, int height, unsigned char *out_data, unsigned char *ou_data)  //out_data:d_result   直接输出四个独立的二值化标记图
{
	int size = width * height;
	hipMemcpy(out_data, d_whitever, sizeof(unsigned char)*size, hipMemcpyDeviceToHost);
	hipMemcpy(ou_data, d_whitehor, sizeof(unsigned char)*size, hipMemcpyDeviceToHost);
	return 1;
}
int cuda_converter::TranslateInput(int width, int height, unsigned char *pImage)        //输入是灰度图Y矩阵,所以不必乘以3
{
	hipMalloc((void**)&d_yuv, sizeof(unsigned char)*width*height);
	hipMemcpy(d_yuv, pImage, sizeof(unsigned char)*width*height, hipMemcpyHostToDevice);

	return 0;
}

///////////////////////////////////////////////////////blob gpu mem
int cuda_converter::TranslateInputB(int width, int height, int *pImage)        //输入是灰度图Y矩阵,所以不必乘以3
{
	hipMalloc((void**)&d_ptr, sizeof(int)*2*width*height);               //已经对d_ptr赋值，所以MallocMemP中没必要再赋值一遍
	hipMemcpy(d_ptr, pImage,  sizeof(int)*2*width*height, hipMemcpyHostToDevice);

	return 0;
}

void cuda_converter::MallocMemP(unsigned int len)  //len=maxlinewidth*num*2 lenn=720*576
{
	int m_ret_len = len;

	//hipMalloc((void**)&d_ptr,    sizeof(unsigned char)* m_ret_len); //output 黑色垂直线标记图
	hipMalloc((void**)&d_outptr, sizeof(int)* m_ret_len); //对应于d_ptr的*chainposition,和d_ptr一样大小
	hipMemset(d_outptr, 0, sizeof(int)*m_ret_len);                              //为了便于从输出中提取信息，chainposition数组应在一开始全0初始化
}


void cuda_converter::MallocMemM(unsigned int len)  //len=maxlinewidth*num*2 lenn=720*576
{
	int m_ret_len = len;
	hipMalloc((void**)&d_maskimg, sizeof(unsigned char)* m_ret_len); //输出：相当于CPU端的maskimage
	hipMemset(d_maskimg, 0, sizeof(unsigned char)*m_ret_len);
}
int cuda_converter::TranslateOutputM(int len, unsigned char *out_data)        //输入是灰度图Y矩阵,所以不必乘以3
{
	int m_ret_len = len;
	hipMemcpy(out_data, d_maskimg, sizeof(unsigned char)*m_ret_len, hipMemcpyDeviceToHost);
	return 1;
}
int cuda_converter::ReleaseMemM()
{

	hipFree(d_maskimg);
	return 1;
}
///////////////////////对应horizontal水平线的并行函数///////////////////////////////////////////////
int cuda_converter::TranslateInputH(int width, int height, int *pImage)        //输入是灰度图Y矩阵,所以不必乘以3
{
	hipMalloc((void**)&d2_ptr, sizeof(int) * 2 * width*height);               //已经对d_ptr赋值，所以MallocMemP中没必要再赋值一遍
	hipMemcpy(d2_ptr, pImage, sizeof(int) * 2 * width*height, hipMemcpyHostToDevice);

	return 1;
}

void cuda_converter::MallocMemH(unsigned int len)
{
	int m_ret_len = len;
	hipMalloc((void**)&d2_outptr, sizeof(int)* m_ret_len); //对应于d_ptr的*chainposition,和d_ptr一样大小
	hipMemset(d2_outptr, 0, sizeof(int)*m_ret_len);

}
int cuda_converter::TranslateOutputH(int len, int *out_data)    //len=maxlinewidth*num*2 lenn=720*576
{
	int m_ret_len = len;
	hipMemcpy(out_data, d2_outptr, sizeof(int)*m_ret_len, hipMemcpyDeviceToHost);
	return 1;
}

int cuda_converter::ReleaseMemH()
{

	hipFree(d2_ptr);
	hipFree(d2_outptr);
	return 1;
}



////////////////////////////////////////////////////////////////////////////////////////////////////////////
int cuda_converter::ReleaseMemP()
{

	hipFree(d_ptr);
	hipFree(d_outptr);
	return 1;
}
int cuda_converter::TranslateOutputP(int len, int *out_data)    //len=maxlinewidth*num*2 lenn=720*576
{
	int m_ret_len = len;
	hipMemcpy(out_data , d_outptr,  sizeof(int)*m_ret_len, hipMemcpyDeviceToHost);
	return 1;
}

__device__ int chaincode(int *blobmask, unsigned char *maskimage, int *chainposition ,int w, int h)           //如果chaincode很复杂，那就必须独立写成一个kernel，这样就关系到d_temp的传输了，从fill()中输出，显存释放函数hipFree可能不能和初始化函数
{


	//在同一个函数体内，这样的话是否会造成矛盾  w:maxlinewidth h:num
	bool burrflag = false;
	//chain code analysis	
	int clock[8][2] = { { 1, 0 }, { 1, -1 }, { 1, -1 }, { 1, -1 }, { 1, -1 }, { -1, 1 }, { 0, 1 }, { 1, 1 } };           //跳过2，3，4方向，避免搜索原路返回
	int anticlock[8][2] = { { 1, 0 }, { 1, 1 }, { 0, 1 }, { -1, 1 }, { -1, 0 }, { -1, -1 }, { 0, -1 }, { 1, -1 } };
	int anticlock1[8][2] = { { -1, 0 }, { -1, 1 }, { 0, 1 }, { 1, 1 }, { 1, 0 }, { 1, -1 }, { 0, -1 }, { -1, -1 } };
	//重排序后的blobmask，startx和starty在起点处，按x为主序排列（符合8邻域内的x的条件 差为1，-1，0）

	//链表跟踪值会超出寄存器容量吗？
	unsigned char *chaincode      = new unsigned char[w];            //如果可以直接new分配存储空间的话，全零初始化显得没有必要，因为有自带的计数器
	unsigned char *clockcode      = new unsigned char[w];
	unsigned char *anticlockcode  = new unsigned char[w];
	unsigned char *anticlockcode1 = new unsigned char[w];
	int *tempmask       = new int[2*w];
	int *antichainmask  = new int[2*w];
	int *antichainmask1 = new int[2*w];


	//这里chainposition因为是作为输出变量，不是函数内变量，所以不用new初始化，直接写入数据即可，而记录方向的clockcode等变量是为了后续的比较长度等等
	// 对应CPU版本中的vector模型，使用thrust库来对应，希望接口不会出问题,因为对应每一个blob，直接移到执行函数内部,因为thrust和fprintf一样是host函数，所以此处用不了

	//因为CPU中用到了vector向量，所以不急着实施，先看看输出图像效果怎样（链表分析函数是否顺利运行）
	int rx, ry, lx, ly, tx, ty, bx, by;  //分别储存四个坐标（八个数值）
	int startx, starty, endx, endy;
	rx = lx = tx = bx = blobmask[0];
	ry = ly = ty = by = blobmask[1];
	for (int i = 0; i < w; i++)
	{
		if ((blobmask[2 * i] > 0) && (blobmask[2 * i + 1] > 0)){
			if (blobmask[2 * i] > rx)
			{
				rx = blobmask[2 * i];
				ry = blobmask[2 * i + 1];
			}
			if (blobmask[2 * i] < lx)
			{
				lx = blobmask[2 * i];
				ly = blobmask[2 * i + 1];
			}
			if (blobmask[2 * i + 1] > ty)
			{
				tx = blobmask[2 * i];
				ty = blobmask[2 * i + 1];
			}
			if (blobmask[2 * i + 1] < by)
			{
				bx = blobmask[2 * i];
				by = blobmask[2 * i + 1];
			}
		}
	}
	if (rx - lx >= ty - by)
	{
		startx = lx;
		starty = ly;
		endx = rx;
		endy = ry;
	}
	else
	{
		startx = bx;
		starty = by;
		endx = tx;
		endy = ty;
	}

	if (lx == WIDTH || ty == HEIGHT)
	{
		startx = 0;
		starty = 0;
		endx = 0;
		endy = 0;
		return -1;
	}
	/*tempmask[0] = startx;
	tempmask[1] = starty;*/

	//c:对应于每一行的blob的实际长度
	//默认前提：i=w-1，m=w-2时，最大blob的终点（blobmask[2*i]）在次点的邻域内（被包含进去）
	int c = 0;
	for (int m = 0; m < w; m++){
		if (blobmask[2 * m] > 0){
			int mx = blobmask[2 * m];
			int my = blobmask[2 * m + 1];//作为被比较的点也得放进tempmask数组中
			tempmask[2 * c] = mx;
			tempmask[2 * c + 1] = my;
			/*int x = tempmask[2 * c];
			int y = tempmask[2 * c + 1];
			maskimage[y*WIDTH + x] = 1;*/
			c++;
		}
	}
	
    //因为退出循环前c还加了一次，所以c应该代表的是元素个数
	for (int p = 0; p < c;p++)
	{
		
			int x = tempmask[2 * p];
			int y = tempmask[2 * p + 1];
			antichainmask[2 * p] = x;
			antichainmask1[2 * p] = x;
			antichainmask[2 * p + 1] = y;
			antichainmask1[2 * p + 1] = y;
			//maskimage[y*WIDTH + x] = 1;
		
	}
	
	int count, anticount, antcount;     //相当于clock.size()记数
	int curx, cury;                    //记录目前搜索的坐标位置
	//初始链码
	/*clockstart.x = start.x;
	clockstart.y = start.y;*/
	if (rx - lx >= ty - by)
	{
		startx = lx;
		starty = ly;
		endx = rx;
		endy = ry;
	}
	else
	{
		startx = bx;
		starty = by;
		endx = tx;
		endy = ty;
	}

	if (lx == WIDTH || ty == HEIGHT)
	{
		startx = 0;
		starty = 0;
		endx = 0;
		endy = 0;
		return -1;
	}
	for (int i = 0; i < c; i++){
		if ((tempmask[2 * i] == startx) && (tempmask[2 * i + 1] == starty)){
			int x = tempmask[2 * i];
			int y = tempmask[2 * i + 1];
			//maskimage[y*WIDTH + x] = 1;
			tempmask[2*i] = 0;
			tempmask[2*i+1] = 0;   //开头的两个都得赋值为0
			break;
		}
	}
	clockcode[0] = 0;
	count = 1;

	for (int q = 0; q < c; q++)         //搜索到的点个数肯定小于c,用最外层的for循环来取代可能造成死循环的do-while结构
	{
		if (burrflag == false){
			for (int k = 0; k < 8; k++)
			{
				burrflag = false;
				curx = startx + clock[k][0];
				cury = starty + clock[k][1];
				if ((curx > -1) && (curx < WIDTH) && (cury > -1) && (cury < HEIGHT))
				{
					for (int i = 0; i < c; i++)
					{
						if ((tempmask[2 * i] == curx) && (tempmask[2 * i + 1] == cury))
						{
							//maskimage[cury*WIDTH + curx] = 1;       //直接测试第一个函数是否能成功
							tempmask[2 * i] = tempmask[2 * i + 1] = 0;
							clockcode[count] = k;
							count++;
							startx = curx;
							starty = cury;
							burrflag = true;
							goto verify;           //break只能跳出最内层的循环，而break总是与if成对出现，来跳出循环
						}
					}
					
				}
			}
	      verify:
	     if (burrflag)
			{
			   int distx = abs(startx - endx);
			   int disty = abs(starty - endy);

				if (distx < 1 && disty < 1)      //(StartP.x == right.x) && (StartP.y == right.y)
					burrflag = true;
				else
					burrflag = false;
			 }
		else
			burrflag = true;
		}
	}
	                   //while (burrflag == false);
	//clockdist = sqrtf((clockstart.x - start.x)*(clockstart.x - start.x) + (clockstart.y - start.y)*(clockstart.y - start.y));

	//step2:逆时针方向搜索链码
	
	if (rx - lx >= ty - by)
	{
		startx = lx;
		starty = ly;
		endx = rx;
		endy = ry;
	}
	else
	{
		startx = bx;
		starty = by;
		endx = tx;
		endy = ty;
	}

	if (lx == WIDTH || ty == HEIGHT)
	{
		startx = 0;
		starty = 0;
		endx = 0;
		endy = 0;
		return -1;
	}
	///*anticlockstart.x = start.x;
	//anticlockstart.y = start.y;*/
	//三个搜索函数必须注意三个变量 clockcode,count,tempmask分别对应改变
	for (int i = 0; i < c; i++){
		if ((antichainmask[2 * i] == startx) && (antichainmask[2 * i + 1] == starty)){
			int x = antichainmask[2 * i];
			int y = antichainmask[2 * i + 1];
			antichainmask[2 * i] = 0;
			antichainmask[2 * i + 1] = 0;   //开头的两个都得赋值为0
			break;
		}
	}
	anticlockcode[0] = 0;
	anticount = 1;
	burrflag = false;
	for (int q = 0; q < c; q++)         //搜索到的点个数肯定小于c,用最外层的for循环来取代可能造成死循环的do-while结构
	{
		if (burrflag == false){
			for (int k = 0; k < 8; k++)
			{
				burrflag = false;
				curx = startx + anticlock[k][0];
				cury = starty + anticlock[k][1];
				if ((curx > -1) && (curx < WIDTH) && (cury > -1) && (cury < HEIGHT))
				{
					for (int i = 0; i < c; i++)
					{
						if ((antichainmask[2 * i] == curx) && (antichainmask[2 * i + 1] == cury))
						{
							//maskimage[cury*WIDTH + curx] = 1;       //直接测试第一个函数是否能成功
							antichainmask[2 * i] = antichainmask[2 * i + 1] = 0;
							anticlockcode[anticount] = k;
							anticount++;
							startx = curx;
							starty = cury;
							burrflag = true;
							goto antiverify;           //break只能跳出最内层的循环，而break总是与if成对出现，来跳出循环
						}
					}

				}
			}
		antiverify:
			if (burrflag)
			{
				int distx = abs(startx - endx);
				int disty = abs(starty - endy);

				if (distx < 1 && disty < 1)      //(StartP.x == right.x) && (StartP.y == right.y)
					burrflag = true;
				else
					burrflag = false;
			}
			else
				burrflag = true;
		}
	}

	//} while (burrflag == false);
	////anticlockdist = sqrtf((anticlockstart.x - start.x)*(anticlockstart.x - start.x) + (anticlockstart.y - start.y)*(anticlockstart.y - start.y));

	////step3:延迟方向搜索链码
	
	if (rx - lx >= ty - by)
	{
		startx = lx;
		starty = ly;
		endx = rx;
		endy = ry;
	}
	else
	{
		startx = bx;
		starty = by;
		endx = tx;
		endy = ty;
	}

	if (lx == WIDTH || ty == HEIGHT)
	{
		startx = 0;
		starty = 0;
		endx = 0;
		endy = 0;
		return -1;
	}
	///*anticlockstart1.x = start.x;
	//anticlockstart1.y = start.y;*/

	//antichainmask1[0] = 0;
	//antichainmask1[1] = 0;
	//anticlockcode1[0] = 0;
	//antcount = 1;
	for (int i = 0; i < c; i++){
		if ((antichainmask1[2 * i] == startx) && (antichainmask1[2 * i + 1] == starty)){
			int x = antichainmask1[2 * i];
			int y = antichainmask1[2 * i + 1];
			antichainmask1[2 * i] = 0;
			antichainmask1[2 * i + 1] = 0;   //开头的两个都得赋值为0
			break;
		}
	}
	anticlockcode1[0] = 0;
	antcount = 1;
	burrflag = false;
	for (int q = 0; q < c; q++)         //搜索到的点个数肯定小于c,用最外层的for循环来取代可能造成死循环的do-while结构
	{
		if (burrflag == false){
			for (int k = 0; k < 8; k++)
			{
				burrflag = false;
				curx = startx + anticlock1[k][0];
				cury = starty + anticlock1[k][1];
				if ((curx > -1) && (curx < WIDTH) && (cury > -1) && (cury < HEIGHT))
				{
					for (int i = 0; i < c; i++)
					{
						if ((antichainmask1[2 * i] == curx) && (antichainmask1[2 * i + 1] == cury))
						{
							//maskimage[cury*WIDTH + curx] = 1;       //直接测试第一个函数是否能成功
							antichainmask1[2 * i] = antichainmask1[2 * i + 1] = 0;
							anticlockcode1[antcount] = k;
							antcount++;
							startx = curx;
							starty = cury;
							burrflag = true;
							goto antverify;           //break只能跳出最内层的循环，而break总是与if成对出现，来跳出循环
						}
					}

				}
			}
		antverify:
			if (burrflag)
			{
				int distx = abs(startx - endx);
				int disty = abs(starty - endy);

				if (distx < 1 && disty < 1)      //(StartP.x == right.x) && (StartP.y == right.y)
					burrflag = true;
				else
					burrflag = false;
			}
			else
				burrflag = true;
		}
	}
	

	//} while (burrflag == false);
	//anticlockdist1 = sqrtf((anticlockstart1.x - start.x)*(anticlockstart1.x - start.x) + (anticlockstart1.y - start.y)*(anticlockstart1.y - start.y));

	//step4:判决最优链码
	int retflag = 1;
	//int clocksize = clockcode.size();	//所有标准库对应的函数都能在thrust中使用？重载是个好东西
	//int anticlocksize = anticlockcode.size();
	//int anticlocksize1 = anticlockcode1.size();

	//fprintf(fp_chain, "chaincode-algorithm-xy  %10d, %10d, %10d, %3.3f\n", clocksize, anticlocksize, anticlocksize1);
	if (rx - lx >= ty - by)
	{
		startx = lx;
		starty = ly;
		endx = rx;
		endy = ry; 
	}
	else
	{
		startx = bx;
		starty = by;
		endx = tx;
		endy = ty;
	}

	if (lx == WIDTH || ty == HEIGHT)
	{
		startx = 0;
		starty = 0;
		endx = 0;
		endy = 0;
		return -1;
	}
	maskimage[(starty)*WIDTH + (startx)] = 1;                //直接通过反坐标系转换映射到大图maskimage中，而不是对应每个blob的小图temp
	chainposition[0] = startx;
	chainposition[1] = starty;
	//不用将起点的坐标写入chainposition吗？
	if (count >= anticount)
	{
		if (count >= antcount)
		{
			retflag = 1;
			for (int k = 0; k < count; k++)
			{
				int jj = clockcode[k];  //at()有边界检查，而operator没有，有边界溢出可能
				chaincode[k] = jj;
				startx = startx + clock[jj][0];
				starty = starty + clock[jj][1];
				chainposition[2 * (k + 1) + 0] = startx;
				chainposition[2 * (k + 1) + 1] = starty;
				maskimage[(starty)*WIDTH + (startx)] = 1; 
				//fprintf(fp_chain, "chaincode-function-xy  %10d,%10d, %10d\n", start.x, start.y, jj);
			}
		}
		else
		{
			retflag = 2;
			for (int k = 0; k < antcount; k++)
			{
				int jj = anticlockcode1[k];
				chaincode[k] = jj;
				startx = startx + anticlock1[jj][0];
				starty = starty + anticlock1[jj][1];
				//chainposition.push_back(start);
				maskimage[(starty)*WIDTH + (startx)] = 2;
				chainposition[2 * (k + 1) + 0] = startx;
				chainposition[2 * (k + 1) + 1] = starty;
			}
		}
	}
	else
	{
		if (anticount >= antcount)
		{
			retflag = 0;
			for (int k = 0; k < anticount; k++)
			{
				int jj = anticlockcode[k];
				chaincode[k] = jj;
				startx = startx + anticlock[jj][0];
				starty = starty + anticlock[jj][1];
				//chainposition.push_back(start);
				maskimage[(starty)*WIDTH + (startx)] = 3;
				chainposition[2 * (k + 1) + 0] = startx;
				chainposition[2 * (k + 1) + 1] = starty;
			}
		}
		else
		{
			retflag = 2;
			for (int k = 0; k < antcount; k++)
			{
				int jj = anticlockcode1[k];
				chaincode[k] = jj;
				startx = startx + anticlock1[jj][0];
				starty = starty + anticlock1[jj][1];
				//chainposition.push_back(start);
				maskimage[(starty)*WIDTH + (startx)] = 2;
				chainposition[2 * (k + 1) + 0] = startx;
				chainposition[2 * (k + 1) + 1] = starty;
			}
		}
	}

	/*clockcode.clear();
	anticlockcode.clear();
	anticlockcode1.clear();
	chaincode.clear();
	chainposition.clear();*/
    delete tempmask;
	delete antichainmask;
	delete antichainmask1;
	delete clockcode;
	delete anticlockcode;
	delete anticlockcode1;
	delete chaincode;        //如果要观察chaincode的话，应当copy保存至一个数组中输出，以便后续传到host端
	return retflag;
}

__global__ void
d_chaincode_global(int *id, unsigned char *maskimage, int *od,int w, int h){
	unsigned int y = blockIdx.x*blockDim.x + threadIdx.x;
	if (y<h){
		chaincode(&id[2 * y*w], maskimage, &od[2 * y*w],w,h);             //因为每一行实际长度为2*maxlinewidth，所以每一行的行首索引应该是2*y*w
	}
}

double cuda_converter::chaincodekernel(int width, int height)
{
	// var for kernel computation timing
	double dKernelTime;

	//make sure input d_rgb was OK

	//maybe donnot need hipDeviceSynchronize() 

	d_chaincode_global <<<  (height + BLOCKSIZE2 - 1) / BLOCKSIZE2, BLOCKSIZE2, 0 >>>(d_ptr, d_maskimg, d_outptr, width, height);

	return dKernelTime;

}
///////////////////////////////////////////////////////////////////////////
__device__ int chaincodeh(int *blobmask, unsigned char *maskimage, int *chainposition, int w, int h)           //如果chaincode很复杂，那就必须独立写成一个kernel，这样就关系到d_temp的传输了，从fill()中输出，显存释放函数hipFree可能不能和初始化函数
{


	//在同一个函数体内，这样的话是否会造成矛盾  w:maxlinewidth h:num
	bool burrflag = false;
	//chain code analysis	
	int clock[8][2] = { { 1, 0 }, { 1, -1 }, { 1, -1 }, { 1, -1 }, { 1, -1 }, { -1, 1 }, { 0, 1 }, { 1, 1 } };           //跳过2，3，4方向，避免搜索原路返回
	int anticlock[8][2] = { { 1, 0 }, { 1, 1 }, { 0, 1 }, { -1, 1 }, { -1, 0 }, { -1, -1 }, { 0, -1 }, { 1, -1 } };
	int anticlock1[8][2] = { { -1, 0 }, { -1, 1 }, { 0, 1 }, { 1, 1 }, { 1, 0 }, { 1, -1 }, { 0, -1 }, { -1, -1 } };
	//重排序后的blobmask，startx和starty在起点处，按x为主序排列（符合8邻域内的x的条件 差为1，-1，0）

	//链表跟踪值会超出寄存器容量吗？
	unsigned char *chaincode = new unsigned char[w];            //如果可以直接new分配存储空间的话，全零初始化显得没有必要，因为有自带的计数器
	unsigned char *clockcode = new unsigned char[w];
	unsigned char *anticlockcode = new unsigned char[w];
	unsigned char *anticlockcode1 = new unsigned char[w];
	int *tempmask = new int[2 * w];
	int *antichainmask = new int[2 * w];
	int *antichainmask1 = new int[2 * w];


	//这里chainposition因为是作为输出变量，不是函数内变量，所以不用new初始化，直接写入数据即可，而记录方向的clockcode等变量是为了后续的比较长度等等
	// 对应CPU版本中的vector模型，使用thrust库来对应，希望接口不会出问题,因为对应每一个blob，直接移到执行函数内部,因为thrust和fprintf一样是host函数，所以此处用不了

	//因为CPU中用到了vector向量，所以不急着实施，先看看输出图像效果怎样（链表分析函数是否顺利运行）
	int rx, ry, lx, ly, tx, ty, bx, by;  //分别储存四个坐标（八个数值）
	int startx, starty, endx, endy;
	rx = lx = tx = bx = blobmask[0];
	ry = ly = ty = by = blobmask[1];
	for (int i = 0; i < w; i++)
	{
		if ((blobmask[2 * i] > 0) && (blobmask[2 * i + 1] > 0)){
			if (blobmask[2 * i] > rx)
			{
				rx = blobmask[2 * i];
				ry = blobmask[2 * i + 1];
			}
			if (blobmask[2 * i] < lx)
			{
				lx = blobmask[2 * i];
				ly = blobmask[2 * i + 1];
			}
			if (blobmask[2 * i + 1] > ty)
			{
				tx = blobmask[2 * i];
				ty = blobmask[2 * i + 1];
			}
			if (blobmask[2 * i + 1] < by)
			{
				bx = blobmask[2 * i];
				by = blobmask[2 * i + 1];
			}
		}
	}
	if (rx - lx >= ty - by)
	{
		startx = lx;
		starty = ly;
		endx = rx;
		endy = ry;
	}
	else
	{
		startx = bx;
		starty = by;
		endx = tx;
		endy = ty;
	}

	if (lx == WIDTH || ty == HEIGHT)
	{
		startx = 0;
		starty = 0;
		endx = 0;
		endy = 0;
		return -1;
	}
	/*tempmask[0] = startx;
	tempmask[1] = starty;*/

	//c:对应于每一行的blob的实际长度
	//默认前提：i=w-1，m=w-2时，最大blob的终点（blobmask[2*i]）在次点的邻域内（被包含进去）
	int c = 0;
	for (int m = 0; m < w; m++){
		if (blobmask[2 * m] > 0){
			int mx = blobmask[2 * m];
			int my = blobmask[2 * m + 1];//作为被比较的点也得放进tempmask数组中
			tempmask[2 * c] = mx;
			tempmask[2 * c + 1] = my;
			/*int x = tempmask[2 * c];
			int y = tempmask[2 * c + 1];
			maskimage[y*WIDTH + x] = 1;*/
			c++;
		}
	}

	//因为退出循环前c还加了一次，所以c应该代表的是元素个数
	for (int p = 0; p < c; p++)
	{

		int x = tempmask[2 * p];
		int y = tempmask[2 * p + 1];
		antichainmask[2 * p] = x;
		antichainmask1[2 * p] = x;
		antichainmask[2 * p + 1] = y;
		antichainmask1[2 * p + 1] = y;
		//maskimage[y*WIDTH + x] = 1;

	}

	int count, anticount, antcount;     //相当于clock.size()记数
	int curx, cury;                    //记录目前搜索的坐标位置
	//初始链码
	/*clockstart.x = start.x;
	clockstart.y = start.y;*/
	if (rx - lx >= ty - by)
	{
		startx = lx;
		starty = ly;
		endx = rx;
		endy = ry;
	}
	else
	{
		startx = bx;
		starty = by;
		endx = tx;
		endy = ty;
	}

	if (lx == WIDTH || ty == HEIGHT)
	{
		startx = 0;
		starty = 0;
		endx = 0;
		endy = 0;
		return -1;
	}
	for (int i = 0; i < c; i++){
		if ((tempmask[2 * i] == startx) && (tempmask[2 * i + 1] == starty)){
			int x = tempmask[2 * i];
			int y = tempmask[2 * i + 1];
			//maskimage[y*WIDTH + x] = 1;
			tempmask[2 * i] = 0;
			tempmask[2 * i + 1] = 0;   //开头的两个都得赋值为0
			break;
		}
	}
	clockcode[0] = 0;
	count = 1;

	for (int q = 0; q < c; q++)         //搜索到的点个数肯定小于c,用最外层的for循环来取代可能造成死循环的do-while结构
	{
		if (burrflag == false){
			for (int k = 0; k < 8; k++)
			{
				burrflag = false;
				curx = startx + clock[k][1];
				cury = starty + clock[k][0];
				if ((curx > -1) && (curx < WIDTH) && (cury > -1) && (cury < HEIGHT))
				{
					for (int i = 0; i < c; i++)
					{
						if ((tempmask[2 * i] == curx) && (tempmask[2 * i + 1] == cury))
						{
							//maskimage[cury*WIDTH + curx] = 1;       //直接测试第一个函数是否能成功
							tempmask[2 * i] = tempmask[2 * i + 1] = 0;
							clockcode[count] = k;
							count++;
							startx = curx;
							starty = cury;
							burrflag = true;
							goto verifyh;           //break只能跳出最内层的循环，而break总是与if成对出现，来跳出循环
						}
					}

				}
			}
		verifyh:
			if (burrflag)
			{
				int distx = abs(startx - endx);
				int disty = abs(starty - endy);

				if (distx < 1 && disty < 1)      //(StartP.x == right.x) && (StartP.y == right.y)
					burrflag = true;
				else
					burrflag = false;
			}
			else
				burrflag = true;
		}
	}
	//while (burrflag == false);
	//clockdist = sqrtf((clockstart.x - start.x)*(clockstart.x - start.x) + (clockstart.y - start.y)*(clockstart.y - start.y));

	//step2:逆时针方向搜索链码

	if (rx - lx >= ty - by)
	{
		startx = lx;
		starty = ly;
		endx = rx;
		endy = ry;
	}
	else
	{
		startx = bx;
		starty = by;
		endx = tx;
		endy = ty;
	}

	if (lx == WIDTH || ty == HEIGHT)
	{
		startx = 0;
		starty = 0;
		endx = 0;
		endy = 0;
		return -1;
	}
	///*anticlockstart.x = start.x;
	//anticlockstart.y = start.y;*/
	//三个搜索函数必须注意三个变量 clockcode,count,tempmask分别对应改变
	for (int i = 0; i < c; i++){
		if ((antichainmask[2 * i] == startx) && (antichainmask[2 * i + 1] == starty)){
			int x = antichainmask[2 * i];
			int y = antichainmask[2 * i + 1];
			antichainmask[2 * i] = 0;
			antichainmask[2 * i + 1] = 0;   //开头的两个都得赋值为0
			break;
		}
	}
	anticlockcode[0] = 0;
	anticount = 1;
	burrflag = false;
	for (int q = 0; q < c; q++)         //搜索到的点个数肯定小于c,用最外层的for循环来取代可能造成死循环的do-while结构
	{
		if (burrflag == false){
			for (int k = 0; k < 8; k++)
			{
				burrflag = false;
				curx = startx + anticlock[k][1];
				cury = starty + anticlock[k][0];
				if ((curx > -1) && (curx < WIDTH) && (cury > -1) && (cury < HEIGHT))
				{
					for (int i = 0; i < c; i++)
					{
						if ((antichainmask[2 * i] == curx) && (antichainmask[2 * i + 1] == cury))
						{
							//maskimage[cury*WIDTH + curx] = 1;       //直接测试第一个函数是否能成功
							antichainmask[2 * i] = antichainmask[2 * i + 1] = 0;
							anticlockcode[anticount] = k;
							anticount++;
							startx = curx;
							starty = cury;
							burrflag = true;
							goto antiverifyh;           //break只能跳出最内层的循环，而break总是与if成对出现，来跳出循环
						}
					}

				}
			}
		antiverifyh:
			if (burrflag)
			{
				int distx = abs(startx - endx);
				int disty = abs(starty - endy);

				if (distx < 1 && disty < 1)      //(StartP.x == right.x) && (StartP.y == right.y)
					burrflag = true;
				else
					burrflag = false;
			}
			else
				burrflag = true;
		}
	}

	//} while (burrflag == false);
	////anticlockdist = sqrtf((anticlockstart.x - start.x)*(anticlockstart.x - start.x) + (anticlockstart.y - start.y)*(anticlockstart.y - start.y));

	////step3:延迟方向搜索链码

	if (rx - lx >= ty - by)
	{
		startx = lx;
		starty = ly;
		endx = rx;
		endy = ry;
	}
	else
	{
		startx = bx;
		starty = by;
		endx = tx;
		endy = ty;
	}

	if (lx == WIDTH || ty == HEIGHT)
	{
		startx = 0;
		starty = 0;
		endx = 0;
		endy = 0;
		return -1;
	}
	///*anticlockstart1.x = start.x;
	//anticlockstart1.y = start.y;*/

	//antichainmask1[0] = 0;
	//antichainmask1[1] = 0;
	//anticlockcode1[0] = 0;
	//antcount = 1;
	for (int i = 0; i < c; i++){
		if ((antichainmask1[2 * i] == startx) && (antichainmask1[2 * i + 1] == starty)){
			int x = antichainmask1[2 * i];
			int y = antichainmask1[2 * i + 1];
			antichainmask1[2 * i] = 0;
			antichainmask1[2 * i + 1] = 0;   //开头的两个都得赋值为0
			break;
		}
	}
	anticlockcode1[0] = 0;
	antcount = 1;
	burrflag = false;
	for (int q = 0; q < c; q++)         //搜索到的点个数肯定小于c,用最外层的for循环来取代可能造成死循环的do-while结构
	{
		if (burrflag == false){
			for (int k = 0; k < 8; k++)
			{
				burrflag = false;
				curx = startx + anticlock1[k][1];
				cury = starty + anticlock1[k][0];
				if ((curx > -1) && (curx < WIDTH) && (cury > -1) && (cury < HEIGHT))
				{
					for (int i = 0; i < c; i++)
					{
						if ((antichainmask1[2 * i] == curx) && (antichainmask1[2 * i + 1] == cury))
						{
							//maskimage[cury*WIDTH + curx] = 1;       //直接测试第一个函数是否能成功
							antichainmask1[2 * i] = antichainmask1[2 * i + 1] = 0;
							anticlockcode1[antcount] = k;
							antcount++;
							startx = curx;
							starty = cury;
							burrflag = true;
							goto antverifyh;           //break只能跳出最内层的循环，而break总是与if成对出现，来跳出循环
						}
					}

				}
			}
		antverifyh:
			if (burrflag)
			{
				int distx = abs(startx - endx);
				int disty = abs(starty - endy);

				if (distx < 1 && disty < 1)      //(StartP.x == right.x) && (StartP.y == right.y)
					burrflag = true;
				else
					burrflag = false;
			}
			else
				burrflag = true;
		}
	}


	//} while (burrflag == false);
	//anticlockdist1 = sqrtf((anticlockstart1.x - start.x)*(anticlockstart1.x - start.x) + (anticlockstart1.y - start.y)*(anticlockstart1.y - start.y));

	//step4:判决最优链码
	int retflag = 1;
	//int clocksize = clockcode.size();	//所有标准库对应的函数都能在thrust中使用？重载是个好东西
	//int anticlocksize = anticlockcode.size();
	//int anticlocksize1 = anticlockcode1.size();

	//fprintf(fp_chain, "chaincode-algorithm-xy  %10d, %10d, %10d, %3.3f\n", clocksize, anticlocksize, anticlocksize1);
	if (rx - lx >= ty - by)
	{
		startx = lx;
		starty = ly;
		endx = rx;
		endy = ry;
	}
	else
	{
		startx = bx;
		starty = by;
		endx = tx;
		endy = ty;
	}

	if (lx == WIDTH || ty == HEIGHT)
	{
		startx = 0;
		starty = 0;
		endx = 0;
		endy = 0;
		return -1;
	}
	maskimage[(starty)*WIDTH + (startx)] = 1;                //直接通过反坐标系转换映射到大图maskimage中，而不是对应每个blob的小图temp
	chainposition[0] = startx;
	chainposition[1] = starty;
	//不用将起点的坐标写入chainposition吗？
	if (count >= anticount)
	{
		if (count >= antcount)
		{
			retflag = 1;
			for (int k = 0; k < count; k++)
			{
				int jj = clockcode[k];  //at()有边界检查，而operator没有，有边界溢出可能
				chaincode[k] = jj;
				startx = startx + clock[jj][1];
				starty = starty + clock[jj][0];
				chainposition[2 * (k + 1) + 0] = startx;
				chainposition[2 * (k + 1) + 1] = starty;
				maskimage[(starty)*WIDTH + (startx)] = 1;
				//fprintf(fp_chain, "chaincode-function-xy  %10d,%10d, %10d\n", start.x, start.y, jj);
			}
		}
		else
		{
			retflag = 2;
			for (int k = 0; k < antcount; k++)
			{
				int jj = anticlockcode1[k];
				chaincode[k] = jj;
				startx = startx + anticlock1[jj][1];
				starty = starty + anticlock1[jj][0];
				//chainposition.push_back(start);
				maskimage[(starty)*WIDTH + (startx)] = 2;
				chainposition[2 * (k + 1) + 0] = startx;
				chainposition[2 * (k + 1) + 1] = starty;
			}
		}
	}
	else
	{
		if (anticount >= antcount)
		{
			retflag = 0;
			for (int k = 0; k < anticount; k++)
			{
				int jj = anticlockcode[k];
				chaincode[k] = jj;
				startx = startx + anticlock[jj][1];
				starty = starty + anticlock[jj][0];
				//chainposition.push_back(start);
				maskimage[(starty)*WIDTH + (startx)] = 3;
				chainposition[2 * (k + 1) + 0] = startx;
				chainposition[2 * (k + 1) + 1] = starty;
			}
		}
		else
		{
			retflag = 2;
			for (int k = 0; k < antcount; k++)
			{
				int jj = anticlockcode1[k];
				chaincode[k] = jj;
				startx = startx + anticlock1[jj][1];
				starty = starty + anticlock1[jj][0];
				//chainposition.push_back(start);
				maskimage[(starty)*WIDTH + (startx)] = 2;
				chainposition[2 * (k + 1) + 0] = startx;
				chainposition[2 * (k + 1) + 1] = starty;
			}
		}
	}

	/*clockcode.clear();
	anticlockcode.clear();
	anticlockcode1.clear();
	chaincode.clear();
	chainposition.clear();*/
	delete tempmask;
	delete antichainmask;
	delete antichainmask1;
	delete clockcode;
	delete anticlockcode;
	delete anticlockcode1;
	delete chaincode;        //如果要观察chaincode的话，应当copy保存至一个数组中输出，以便后续传到host端
	return retflag;
}

__global__ void
d2_chaincode_global(int *id, unsigned char *maskimage, int *od, int w, int h){
	unsigned int y = blockIdx.x*blockDim.x + threadIdx.x;
	if (y<h){
		chaincodeh(&id[2 * y*w], maskimage, &od[2 * y*w], w, h);             //因为每一行实际长度为2*maxlinewidth，所以每一行的行首索引应该是2*y*w
	}
}

double cuda_converter::chaincodehkernel(int width, int height)
{
	// var for kernel computation timing
	double dKernelTime;

	//make sure input d_rgb was OK

	//maybe donnot need hipDeviceSynchronize() 

	d2_chaincode_global <<<  (height + BLOCKSIZE2 - 1) / BLOCKSIZE2, BLOCKSIZE2, 0 >>>(d2_ptr, d_maskimg, d2_outptr, width, height);

	return dKernelTime;

}

////////////////////////////////////////////////////////////////////////
void cuda_converter::MallocMemA(unsigned int len)  //len=width*height,ip:h_yuv  
{
	m_ret_len = len;


	hipMalloc((void**)&d_blackver, sizeof(unsigned char)* m_ret_len); //output 黑色垂直线标记图
	hipMalloc((void**)&d_blackhor, sizeof(unsigned char)* m_ret_len);
	hipMalloc((void**)&d_whitever, sizeof(unsigned char)* m_ret_len); //output 黑色垂直线标记图
	hipMalloc((void**)&d_whitehor, sizeof(unsigned char)* m_ret_len);

}

int cuda_converter::ReleaseMem()
{

	hipFree(d_yuv);
	hipFree(d_blackver);
	hipFree(d_blackhor);
	hipFree(d_whitever);
	hipFree(d_whitehor);
	return 1;
}


__device__ void
d_markblack_ver(unsigned char *id, unsigned char *od, int w, int h, int msize)
{
	int Y0, s1, s2, sn, snedge, sum1, sum2, sntemp1, sntemp2;
	int T = 4;
	int masksize = msize;
	for (int j = 0; j<masksize / 2; j++)
	{
		od[j] = 0;
	}
	for (int j2 = w - masksize / 2; j2<w; j2++)
	{
		od[j2] = 0;
	}
	for (int i = masksize / 2; i<w - masksize / 2; i++)
	{
		Y0 = id[i];
		s1 = 0;
		s2 = 0;
		sum1 = 0;
		sum2 = 0;
		sntemp1 = 0;
		sntemp2 = 0;

		for (int k = -masksize / 2; k<0; k++)
		{
			s1 = s1 + id[i + masksize / 2] - Y0;
			s2 = s2 + id[i - masksize / 2] - Y0;
			sum1 = sum1 + id[i + masksize / 2];
			sum2 = sum2 + id[i - masksize / 2];

			if (abs(id[i + masksize / 2] - Y0) <= T)
			{
				sntemp1 = sntemp1 + 1;
			}
			if (abs(id[i - masksize / 2] - Y0) <= T)
			{
				sntemp1 = sntemp1 + 1;
			}
			if (abs(id[i + masksize / 2] - Y0) <= T)
			{
				sntemp2 = sntemp2 + 1;
			}
			if (abs(id[i - masksize / 2] - Y0) <= T)
			{
				sntemp2 = sntemp2 + 1;
			}
		}
		sn = sntemp1;
		snedge = sntemp2;

		if (s1>0 && s2>0 && sn<masksize / 2)         //线对象
		{
			od[i] = 1;
		}

		else
		{
			od[i] = 0;
		}
	}





}



__device__ void
d_markblack_hor(unsigned char *id, unsigned char *od, int w, int h, int msize)
{
	int Y0, s1, s2, sn, snedge, sum1, sum2, sntemp1, sntemp2;
	int T = 4;
	int masksize = msize;
	for (int j = 0; j<masksize / 2; j++)
	{
		od[j*w] = 0;
	}
	for (int s = h - masksize / 2; s<h; s++)
	{
		od[s*w] = 0;
	}
	for (int i = masksize / 2; i<h - masksize / 2; i++)
	{
		Y0 = id[i*w];
		s1 = 0;
		s2 = 0;
		sum1 = 0;
		sum2 = 0;
		sntemp1 = 0;
		sntemp2 = 0;

		for (int k = -masksize / 2; k<0; k++)
		{
			s1 = s1 + id[(i + masksize / 2)*w] - Y0;
			s2 = s2 + id[(i - masksize / 2)*w] - Y0;
			sum1 = sum1 + id[(i + masksize / 2)*w];
			sum2 = sum2 + id[(i - masksize / 2)*w];

			if (abs(id[(i + masksize / 2)*w] - Y0) <= T)
			{
				sntemp1 = sntemp1 + 1;
			}
			if (abs(id[(i - masksize / 2)*w] - Y0) <= T)
			{
				sntemp1 = sntemp1 + 1;
			}
			if (abs(id[(i + masksize / 2)*w] - Y0) <= T)
			{
				sntemp2 = sntemp2 + 1;
			}
			if (abs(id[(i - masksize / 2)*w] - Y0) <= T)
			{
				sntemp2 = sntemp2 + 1;
			}
		}
		sn = sntemp1;
		snedge = sntemp2;

		if (s1>0 && s2>0 && sn<masksize / 2)         //线对象
		{
			od[i*w] = 1;
		}

		else
		{
			od[i*w] = 0;
		}
	}





}

__device__ void
d_markwhite_ver(unsigned char *id, unsigned char *od, int w, int h, int msize)
{
	int Y0, s1, s2, sn, snedge, sum1, sum2, sntemp1, sntemp2;
	int T = 4;
	int masksize = msize;
	for (int j = 0; j<masksize / 2; j++)
	{
		od[j] = 0;
	}
	for (int j2 = w - masksize / 2; j2<w; j2++)
	{
		od[j2] = 0;
	}
	for (int i = masksize / 2; i<w - masksize / 2; i++)
	{
		Y0 = id[i];
		s1 = 0;
		s2 = 0;
		sum1 = 0;
		sum2 = 0;
		sntemp1 = 0;
		sntemp2 = 0;

		for (int k = -masksize / 2; k<0; k++)
		{
			s1 = s1 - id[i + masksize / 2] + Y0;
			s2 = s2 - id[i - masksize / 2] + Y0;
			sum1 = sum1 + id[i + masksize / 2];
			sum2 = sum2 + id[i - masksize / 2];

			if (abs(id[i + masksize / 2] - Y0) <= T)
			{
				sntemp1 = sntemp1 + 1;
			}
			if (abs(id[i - masksize / 2] - Y0) <= T)
			{
				sntemp1 = sntemp1 + 1;
			}
			if (abs(id[i + masksize / 2] - Y0) <= T)
			{
				sntemp2 = sntemp2 + 1;
			}
			if (abs(id[i - masksize / 2] - Y0) <= T)
			{
				sntemp2 = sntemp2 + 1;
			}
		}
		sn = sntemp1;
		snedge = sntemp2;

		if (s1>0 && s2>0 && sn<masksize / 2)         //线对象
		{
			od[i] = 1;
		}

		else
		{
			od[i] = 0;
		}
	}





}
__device__ void
d_markwhite_hor(unsigned char *id, unsigned char *od, int w, int h, int msize)
{
	int Y0, s1, s2, sn, snedge, sum1, sum2, sntemp1, sntemp2;
	int T = 4;
	int masksize = msize;
	for (int j = 0; j<masksize / 2; j++)
	{
		od[j*w] = 0;
	}
	for (int s = h - masksize / 2; s<h; s++)
	{
		od[s*w] = 0;
	}
	for (int i = masksize / 2; i<h - masksize / 2; i++)
	{
		Y0 = id[i*w];
		s1 = 0;
		s2 = 0;
		sum1 = 0;
		sum2 = 0;
		sntemp1 = 0;
		sntemp2 = 0;

		for (int k = -masksize / 2; k<0; k++)
		{
			s1 = s1 - id[(i + masksize / 2)*w] + Y0;
			s2 = s2 - id[(i - masksize / 2)*w] + Y0;
			sum1 = sum1 + id[(i + masksize / 2)*w];
			sum2 = sum2 + id[(i - masksize / 2)*w];

			if (abs(id[(i + masksize / 2)*w] - Y0) <= T)
			{
				sntemp1 = sntemp1 + 1;
			}
			if (abs(id[(i - masksize / 2)*w] - Y0) <= T)
			{
				sntemp1 = sntemp1 + 1;
			}
			if (abs(id[(i + masksize / 2)*w] - Y0) <= T)
			{
				sntemp2 = sntemp2 + 1;
			}
			if (abs(id[(i - masksize / 2)*w] - Y0) <= T)
			{
				sntemp2 = sntemp2 + 1;
			}
		}
		sn = sntemp1;
		snedge = sntemp2;

		if (s1>0 && s2>0 && sn<masksize / 2)         //线对象
		{
			od[i*w] = 1;
		}

		else
		{
			od[i*w] = 0;
		}
	}





}

__global__ void
d_markblack_ver_global(unsigned char *id, unsigned char *od, int w, int h, int msize){
	unsigned int y = blockIdx.x*blockDim.x + threadIdx.x;
	d_markblack_ver(&id[y * w], &od[y * w], w, h, msize);

}

__global__ void
d_markblack_hor_global(unsigned char *id, unsigned char *od, int w, int h, int msize){
	unsigned int x = blockIdx.x*blockDim.x + threadIdx.x;
	d_markblack_hor(&id[x], &od[x], w, h, msize);

}

__global__ void
d_markwhite_ver_global(unsigned char *id, unsigned char *od, int w, int h, int msize){
	unsigned int y = blockIdx.x*blockDim.x + threadIdx.x;
	d_markwhite_ver(&id[y * w], &od[y * w], w, h, msize);

}

__global__ void
d_markwhite_hor_global(unsigned char *id, unsigned char *od, int w, int h, int msize){
	unsigned int x = blockIdx.x*blockDim.x + threadIdx.x;
	d_markwhite_hor(&id[x], &od[x], w, h, msize);

}

////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////

//__global__ void mark_blackver(unsigned char *ip, unsigned char *rd, int w, int h, int msize) //img:d_yuv(从host传输过来的Y) rd:(BlackHorEdge) 直接作为输出 
//
//{
//	int i, j, Y0, s1, s2, sn, snedge, sum1, sum2, sntemp1, sntemp2;
//	int T = 4;
//	int masksize = msize;
//
//	//	int masksize=13;                
//	//	int masksize2=9;
//
//	unsigned char *R = NULL;        //用于标记的像素点矩阵
//	unsigned char *Y = NULL;        //亮度矩阵
//
//
//	R = rd;
//	Y = ip;
//
//
//
//
//	uint col = blockIdx.x*blockDim.x + threadIdx.x;          //x
//	uint row = blockIdx.y*blockDim.y + threadIdx.y;          //y
//	uint tim = row*w + col;
//	unsigned char *buff;
//	buff = new unsigned char[w*1];
//	//输入
//	memcpy(buff, Y[tim], w);
//
//	for (int j = 0; j < w; j++)
//	{
//		if (buff[j] > 127)
//			buff[i] = 0;
//		else
//			buff[i] = 1;
//	}
//	memcpy(R[tim], buff, w);
//
//
//	//if ((col < w ) && (row < h ))
//	//{
//	//	if (Y[tim]>127)
//	//	{
//	//		R[tim] = 0;
//	//	}
//	//	else
//	//	{
//	//		R[tim] = 1;
//	//	}
//	//	s1 = 0;
//	//	s2 = 0;
//	//	sum1 = 0;
//	//	sum2 = 0;
//	//	sntemp1 = 0;
//	//	sntemp2 = 0;
//
//	//	for (int k = -masksize / 2; k < 0; k++)
//	//	{
//	//		s1 = s1 + Y[(row)*w + col + masksize / 2] - Y0;           //采用水平模板，同一行的不同列
//	//		s2 = s2 + Y[(row)*w + col - masksize / 2] - Y0;
//	//		sum1 = sum1 + Y[(row)*w + col + masksize / 2];
//	//		sum2 = sum2 + Y[(row)*w + col - masksize / 2];
//
//	//		if (abs(Y[(row)*w + col + masksize / 2] - Y0) <= T)
//	//		{
//	//			sntemp1 = sntemp1 + 1;
//	//		}
//	//		if (abs(Y[(row)*w + col - masksize / 2] - Y0) <= T)
//	//		{
//	//			sntemp1 = sntemp1 + 1;
//	//		}
//	//		if (abs(Y[(row)*w + col + masksize / 2] - Y0) <= T)
//	//		{
//	//			sntemp2 = sntemp2 + 1;
//	//		}
//	//		if (abs(Y[(row)*w + col - masksize / 2] - Y0) <= T)
//	//		{
//	//			sntemp2 = sntemp2 + 1;
//	//		}
//	//	}
//	//	sn = sntemp1;
//	//	snedge = sntemp2;
//
//	//	if (s1 > 0 && s2 > 0 && sn<masksize / 2)         //条带对象
//	//	{
//	//		R[row*w + col] = 1;
//	//	}
//	//	
//	//	else
//	//	{
//	//		R[row*w + col] = 0;
//	//	}
//
//
//
//	}
//
//}







/////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////

double cuda_converter::blackverkernel(int width, int height)
{
	// var for kernel computation timing
	double dKernelTime;

	//make sure input d_rgb was OK

	//maybe donnot need hipDeviceSynchronize() 

	d_markblack_ver_global <<<  height / BLOCKSIZE1, BLOCKSIZE1, 0 >>>(d_yuv, d_blackver, width, height, 9);
	d_markblack_hor_global <<<  width / BLOCKSIZE2, BLOCKSIZE2, 0 >>>(d_yuv, d_blackhor, width, height, 9);

	return dKernelTime;

}

double cuda_converter::whiteverkernel(int width, int height)
{
	// var for kernel computation timing
	double dKernelTime;

	//make sure input d_rgb was OK

	//maybe donnot need hipDeviceSynchronize() 

	d_markwhite_ver_global <<<  height / BLOCKSIZE1, BLOCKSIZE1, 0 >>>(d_yuv, d_whitever, width, height, 9);
	d_markwhite_hor_global <<<  width / BLOCKSIZE2, BLOCKSIZE2, 0 >>>(d_yuv, d_whitehor, width, height, 9);

	return dKernelTime;

}































